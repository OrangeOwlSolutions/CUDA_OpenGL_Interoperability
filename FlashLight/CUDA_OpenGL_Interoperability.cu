#include "hip/hip_runtime.h"
#include "CUDA_Processing.cuh"
#include <stdio.h>
#include <stdlib.h>
#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#include <GL\glew.h>
#include <GL\freeglut.h>
#elif __linux__
#include <GL/glew.h>
#include <GL/freeglut.h>
#else // --- APPLE
#include <GLUT/glut.h>
#endif

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "OpenGL_Keyboard_Mouse.h"

// texture and pixel objects
GLuint pbo = 0;     // OpenGL pixel buffer object
GLuint tex = 0;     // OpenGL texture object
struct hipGraphicsResource *cuda_pbo_resource;

/*******************/
/* RENDER FUNCTION */
/*******************/
// --- Computes new pixel values launching the CUDA kernel
void render() {
	uchar4 *d_out = 0;
	hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
	hipGraphicsResourceGetMappedPointer((void **)&d_out, NULL, cuda_pbo_resource);
	kernelLauncher(d_out, W, H, loc);
	hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
}

/************************/
/* DRAWTEXTURE FUNCTION */
/************************/
// --- Sets up a 2D OpenGL texture image, creates a single quadrangle graphics primitive with
//     texture coordinates (0.0f, 0.0f), (0.0f, 1.0f), (1.0f, 1.0f), and (1.0f, 0.0f); that is,
//     the corners of the unit square, corresponding with the pixel coordinates (0, 0), (0, H), (W, H),
//     and (W, 0).
void drawTexture() {
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, W, H, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
	glEnable(GL_TEXTURE_2D);
	glBegin(GL_QUADS);
	glTexCoord2f(0.0f, 0.0f); glVertex2f(0, 0);
	glTexCoord2f(0.0f, 1.0f); glVertex2f(0, H);
	glTexCoord2f(1.0f, 1.0f); glVertex2f(W, H);
	glTexCoord2f(1.0f, 0.0f); glVertex2f(W, 0);
	glEnd();
	glDisable(GL_TEXTURE_2D);
}

/********************/
/* DISPLAY FUNCTION */
/********************/
void display() {
	render();				// --- Computes new pixel values
	drawTexture();			// --- Draws the OpenGL texture
	glutSwapBuffers();	// --- Swap the read/write buffers
}

/********************************/
/* GLUT INITIALIZATION FUNCTION */
/********************************/
void initGLUT(int *argc, char **argv) {
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(W, H);
	glutCreateWindow(TITLE_STRING);
#ifndef __APPLE__
	glewInit();
#endif
}

/****************************/
/* INITPIXELBUFFER FUNCTION */
/****************************/
// --- Initializes the pixel buffer.
void initPixelBuffer() {
	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, 4 * W*H*sizeof(GLubyte), 0,
		GL_STREAM_DRAW);
	glGenTextures(1, &tex);
	glBindTexture(GL_TEXTURE_2D, tex);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	// --- \u201cRegisters\u201d the OpenGL buffer with CUDA.
	hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard);
}

/*****************/
/* EXIT FUNCTION */
/*****************/
// --- Undoes the resource registration and deletes the OpenGL pixel buffer and texture before zero is
//     returned to indicate completion of main().
void exitfunc() {
	if (pbo) {
		hipGraphicsUnregisterResource(cuda_pbo_resource);
		glDeleteBuffers(1, &pbo);
		glDeleteTextures(1, &tex);
	}
}

/********/
/* MAIN */
/********/
int main(int argc, char** argv) {
	printInstructions();
	// --- Initializes the GLUT library and sets up the specifications for the graphics window,
	//     including the display mode (RGBA), the buffering (double), size (W x H), and title.
	initGLUT(&argc, argv);
	gluOrtho2D(0, W, H, 0);					// --- Establishes the viewing transform (simple orthographic projection)
	glutKeyboardFunc(keyboard);				// --- Keyboard interactions are specified by the function keyboard
	glutSpecialFunc(handleSpecialKeypress);	// --- Special keyboard interactions are specified by the function handleSpecialKeypress
	glutPassiveMotionFunc(mouseMove);			// --- Mouse move interactions are specified by the function mouseMove
	glutMotionFunc(mouseDrag);					// --- Mouse drag interactions are specified by the function mouseMove
	glutDisplayFunc(display);					// --- Says that what is to be shown in the window is determined by the function display
	initPixelBuffer();							// --- Initializes the pixel buffer
	glutMainLoop();							// --- Repeatedly checks for input and calls for computation of updated images
	atexit(exitfunc);							// --- Final clean up
	return 0;
}
