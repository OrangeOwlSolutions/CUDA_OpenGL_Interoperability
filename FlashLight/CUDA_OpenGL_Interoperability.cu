#include "hip/hip_runtime.h"
#ifndef INTERACTIONS_H
#define INTERACTIONS_H
#define W 600					// --- Image width
#define H 600					// --- Image height
#define DELTA 5 				// --- Pixel increment for arrow keys
#define TITLE_STRING "flashlight: distance image display app"
int2 loc = {W/2, H/2};			// --- Initial reference location at {W/2, H/2}, the center of the image.
bool dragMode = false; // mouse tracking mode

/**********************************/
/* KEYBOARD INTERACTIONS FUNCTION */
/**********************************/
void keyboard(unsigned char key, int x, int y) {
	if (key == 'a') dragMode = !dragMode; // --- Pressing a toggles between tracking mouse motions and
										  //     tracking mouse drags (with the mouse button pressed),
	if (key == 27)  exit(0);			  // --- The ASCII code 27 corresponds to the Esc key. Pressing Esc 
										  //     closes the graphics window.
	glutPostRedisplay();				  // --- glutPostRedisplay() is called at the end of each callback
										  //     function telling to compute a new image for display
	                                      //     (by calling the display function) based on the interactive
	                                      //     input
}

/******************************************/
/* SPECIAL KEYBOARD INTERACTIONS FUNCTION */
/******************************************/
// --- Specifies the response to special keys with defined actions (arrow keys)
//     Sepressing the arrow keys moves the reference location DELTA pixels in the desired direction.
void handleSpecialKeypress(int key, int x, int y) {
	if (key == GLUT_KEY_LEFT)  loc.x -= DELTA;
	if (key == GLUT_KEY_RIGHT) loc.x += DELTA;
	if (key == GLUT_KEY_UP)    loc.y -= DELTA;
	if (key == GLUT_KEY_DOWN)  loc.y += DELTA;
	glutPostRedisplay();
}

/************************************/
/* MOUSE MOVE INTERACTIONS FUNCTION */
/************************************/
void mouseMove(int x, int y) {
	if (dragMode) return;				  // --- When dragMode is true, no action is taken
	loc.x = x;							  // --- When dragMode is false, the components of the reference
	                                      //     location are set to be equal to the x and y coordinates of the
	                                      //     mouse before computing and displaying an updated image
	                                      //     (via glutPostRedisplay()).
	loc.y = y;
	glutPostRedisplay();				  // --- See keyboard function
}

/************************************/
/* MOUSE DRAG INTERACTIONS FUNCTION */
/************************************/
void mouseDrag(int x, int y) {
	if (!dragMode) return;				  // --- When dragMode is false, no action is taken
	loc.x = x;							  // --- When dragMode is true, the reference location is reset to
										  //     the last location of the mouse while the mouse was clicked
	loc.y = y;
	glutPostRedisplay();				  // --- See keyboard function
}

/******************************/
/* PRINTINSTRUCTIONS FUNCTION */
/******************************/
// --- Prints instructions through the console
void printInstructions() {
  printf("flashlight interactions\n");
  printf("a: toggle mouse tracking mode\n");
  printf("arrow keys: move ref location\n");
  printf("esc: close graphics window\n");
}

#endif
